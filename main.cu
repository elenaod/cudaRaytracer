#include "hip/hip_runtime.h"
#include <SDL/SDL.h>
#include <utils/sdl.cuh>
#include <kernels.cuh>
#include <cstdio>

SDL_Surface* screen = NULL;

int main(int argc, char** argv) {
  clock_t init_start, init_end, draw_start, draw_end;
  float time;
  init_start = clock();

  const int __SIZE = VFB_MAX_SIZE * VFB_MAX_SIZE;
  Color *host_vfb, *device_vfb;
  bool *needsAA;
  host_vfb = (Color*)malloc(__SIZE * sizeof(Color));
  hipMalloc((void**)&device_vfb, __SIZE * sizeof(Color));
  hipMemcpy(device_vfb,
             host_vfb,
             __SIZE * sizeof(Color),
             hipMemcpyHostToDevice);
  hipMalloc((void**)&needsAA, __SIZE * sizeof(bool));

  Scene *scene = new Scene, *dev_scene;
  if (!initGraphics(&screen, RESX, RESY)) return -1;
  scene->initialize();
  hipMalloc((void**)&dev_scene, sizeof(Scene));
  hipMemcpy(dev_scene, scene, sizeof(Scene), hipMemcpyHostToDevice);

  init_end = clock();
  time = ((float)init_end - (float)init_start) / CLOCKS_PER_SEC;
  printf("Sequential: %f s\n", time);
  draw_start = clock();
  renderScene<<<1, 25>>>(dev_scene, device_vfb);
  findAA<<<1, 25>>>(needsAA, device_vfb);
  antialias<<<1, 25>>>(dev_scene, needsAA, device_vfb);
  hipError_t cudaerr = hipDeviceSynchronize();
  draw_end = clock();
  time = ((float) draw_end - (float)draw_start) / CLOCKS_PER_SEC;
  printf("Parallel on N threads: %f s\n", time);

  hipMemcpy(host_vfb,
            device_vfb,
             __SIZE * sizeof(Color),
             hipMemcpyDeviceToHost);
  displayVFB(screen, host_vfb);
  SDL_SaveBMP(screen, "output.bmp");
  waitForUserExit();
  closeGraphics();

  scene->cleanUp();
  free(scene); hipFree(dev_scene);
  free(host_vfb);
  hipFree(device_vfb);
  hipFree(needsAA);

  return 0;
}
