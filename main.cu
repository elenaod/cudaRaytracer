#include "hip/hip_runtime.h"
#include <SDL/SDL.h>
#include <thrust/device_vector.h>
#include "sdl.cuh"
#include "matrix.cuh"
#include "camera.cuh"
#include "geometry.cuh"
#include "shading.cuh"
#include "cuPrintf.cu"
using namespace std;

#include <cstdio>

// remove global variables
// come to think of it, separate struct for the scene?
// I need the camera and the nodes everywhere...

__device__
Color raytrace(Ray ray,
               const Light& _light,
               thrust::device_vector<Node>::iterator start,
               thrust::device_vector<Node>::iterator end){
  IntersectionData data;

  Node value = *start;
  Plane *pl = (Plane*) value.geom;
  printf("raytrace::pl = %llu, pl->y = %d\n", pl, pl->y);
  pl->intersect(ray, data);
  printf("Y");

  for (thrust::device_vector<Node>::iterator iter = start;
         iter != end; ++iter){
    Node value = *iter;
    if (value.geom->intersect(ray, data)){
////      return value.shader->shade(ray, _light, data);
      return Color (0,1,0);
    }
  }
  return Color(0, 0, 0);
}

__global__
void debugPrint(Geometry *plane){
  printf("=== debug print ===\n");
  Plane *pl = (Plane*) plane;
  printf("plane %llu, %d\n", pl, pl->y);
  printf("=== debug print ===\n");
}

// makes scene == camera + geometries + shaders + lights
void initializeScene(Camera*& _camera,
                     Light* _light,
                     thrust::device_vector<Geometry*>& _geometries,
                     thrust::device_vector<Shader*>& _shaders,
                     thrust::device_vector<Node>& _nodes) {
  _camera = new Camera;
  _camera->yaw = 0;
  _camera->pitch = -30;
  _camera->roll = 0;
  _camera->fov = 90;
  _camera->aspect = 4. / 3.0;
  _camera->pos = Vector(0,165,0);

  _camera->beginFrame();

  _light->pos = Vector(-30, 100, 250);
  _light->color = Color(1, 1, 1);
  _light->power = 50000;

  Plane* plane = new Plane(2);
  Plane *dev_plane = 0;
  hipMalloc((void**)&dev_plane, sizeof(Plane));
  hipMemcpy(dev_plane, plane, sizeof(Plane), hipMemcpyHostToDevice);
  free(plane);
  _geometries.push_back(dev_plane);

  CheckerShader* checker = new CheckerShader(Color(0, 0, 0),
                                             Color(0, 0.5, 1), 5);
  CheckerShader* dev_checker = 0;
  hipMalloc((void**)&dev_checker, sizeof(CheckerShader));
  hipMemcpy(dev_checker, checker,
             sizeof(CheckerShader), 
             hipMemcpyHostToDevice);
  free(checker);
  _shaders.push_back(dev_checker);

  printf("allocating floor node\n");
  Node floor;
  floor.geom = dev_plane; floor.shader = dev_checker;
  _nodes.push_back(floor);
}

__global__
void renderScene(const Camera& _camera,
                 const Light& _light,
                 thrust::device_vector<Node>::iterator start,
                 thrust::device_vector<Node>::iterator end,
                 Color* buffer) {
  // calculate thread idx
  int idx_thrd_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_thrd_y = blockIdx.y * blockDim.y + threadIdx.y;
  int grid_width = gridDim.x * blockDim.x;
  int idx_thread = idx_thrd_y * grid_width + idx_thrd_x;

  // calculate coordinates of pixel we're painting
  // remove constants
  int x = idx_thread / 4;
  int y = idx_thread % 4;
  for(int i = x * 160; i < (x + 1) * 160; ++i)
    for(int j = y * 160; j < (y + 1) * 160; ++j){
    Ray ray = _camera.getScreenRay(i, j);
    buffer[j * RESX + i] = raytrace(ray, _light, start, end);
  }

  printf("renderScene::Scene rendered\n");
}

int main(int argc, char** argv) {
  const int __PIX = RESX;
  const int __SIZE = __PIX * __PIX;
  Color *host_vfb, *device_vfb;
  SDL_Surface* screen = NULL;

  // get vfb on host
  host_vfb = (Color*)malloc(__SIZE * sizeof(Color));
  for(int i = 0; i < __PIX; ++i){
    for (int j = 0; j < __PIX; ++j)
      host_vfb[i * __PIX + j] = Color (1, 0, 0);
  }

  // get vfb on device
  hipMalloc((void**)&device_vfb, __SIZE * sizeof(Color));
  hipMemcpy(device_vfb,
             host_vfb,
             __SIZE * sizeof(Color),
             hipMemcpyHostToDevice);
  printf("Program start...\n");

  // now those are no the host, originally!
  Camera *camera = 0;
  Light pointLight;
  thrust::device_vector<Geometry*> geometries;
  thrust::device_vector<Shader*> shaders;
  thrust::device_vector<Node> nodes;

  printf("Variables declared...\n");
  if (!initGraphics(&screen, RESX, RESY)) return -1;
  printf("Graphics initialized...\n");
  initializeScene(camera, &pointLight, geometries, shaders, nodes);
  printf("Scene initialized... camera = %d\n", camera);
  printf("Scene initialized... light color = (%f, %f, %f)\n",
      pointLight.color.r, pointLight.color.g, pointLight.color.b);
  printf("Scene initialized... light power = %f\n", pointLight.power);

  printf("Scene initialized... nodes.size: %llu\n", nodes.size());
  printf("Scene initialized... start - end = %llu\n",
           nodes.end() - nodes.begin());
  Camera *device_camera = 0;
  hipMalloc((void**) &device_camera, sizeof(Camera));
  thrust::device_vector<Node>::iterator start = nodes.begin();
  thrust::device_vector<Node>::iterator end = nodes.end();

  printf("Scene initialized... start - end with vars = %llu\n",
           end - start);

  renderScene<<<1, 16>>>(*device_camera, pointLight,
                        start, end, device_vfb);

  hipMemcpy(host_vfb,
             device_vfb,
             __SIZE * sizeof(Color),
             hipMemcpyDeviceToHost);
  printf("Scene rendered... \n");
  displayVFB(screen, host_vfb);
  // remove so we can time
  waitForUserExit();
  printf("Closing graphics...\n");
  // illegal memory access was encountered
  closeGraphics();
  printf("All done, only destructors remain...\n");
  // aand, free!
  free(device_camera);
  delete camera;
  free(host_vfb);
  hipFree(device_vfb);
  return 0;
}
