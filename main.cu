#include "hip/hip_runtime.h"
#include <SDL/SDL.h>
#include <utils/sdl.cuh>
#include <kernels.cuh>
#include <cstdio>

const int __SIZE = VFB_MAX_SIZE * VFB_MAX_SIZE;
SDL_Surface* screen = NULL;

int main(int argc, char** argv) {
  clock_t init_start, init_end, draw_start, draw_end;
  float time;
  unsigned threadCount, numBlocks;

  if (argc < 3) {
    printf("You need to specify the number of threads\n");
    printf("raytracer [threadCount] [numBlocks], by default numBlocks == 1");
  }
  else {
    threadCount = atoi(argv[1]);
    numBlocks = atoi(argv[2]);
  }
  
  setBuckets(threadCount, numBlocks);
  init_start = clock();
  Color *host_vfb, *device_vfb;
  host_vfb = (Color*)malloc(__SIZE * sizeof(Color));
  hipMalloc((void**)&device_vfb, __SIZE * sizeof(Color));
  hipMemcpy(device_vfb, host_vfb, __SIZE * sizeof(Color),
             hipMemcpyHostToDevice);

  bool *needsAA;
  hipMalloc((void**)&needsAA, __SIZE * sizeof(bool));

  Scene *scene = new Scene, *dev_scene;
  if (!initGraphics(&screen, RESX, RESY)) return -1;
  scene->initialize();
  hipMalloc((void**)&dev_scene, sizeof(Scene));
  hipMemcpy(dev_scene, scene, sizeof(Scene), hipMemcpyHostToDevice);

  init_end = clock();
  time = ((float)init_end - (float)init_start) / CLOCKS_PER_SEC;
  printf("Sequential: %f s\n", time);

  draw_start = clock();
  renderScene<<<1, threadCount>>>(dev_scene, device_vfb);
  findAA<<<1, threadCount>>>(needsAA, device_vfb);
  antialias<<<1, threadCount>>>(dev_scene, needsAA, device_vfb);
  hipError_t cudaerr = hipDeviceSynchronize();
  draw_end = clock();
  time = ((float) draw_end - (float)draw_start) / CLOCKS_PER_SEC;
  printf("Parallel on %d threads: %f s\n", threadCount, time);

  hipMemcpy(host_vfb, device_vfb, __SIZE * sizeof(Color),
             hipMemcpyDeviceToHost);
  displayVFB(screen, host_vfb);
  if (argc >= 4) {
    SDL_SaveBMP(screen, argv[3]);
  }

  waitForUserExit();

  closeGraphics();

  scene->cleanUp();
  free(scene); hipFree(dev_scene);
  free(host_vfb);
  hipFree(device_vfb);
  hipFree(needsAA);

  return 0;
}
