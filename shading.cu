#include "hip/hip_runtime.h"
#include "shading.cuh"

Shader::Shader(const Color& color)
{
  this->color = color;
}

CheckerShader::CheckerShader(const Color& c1,
                             const Color& c2,
                             double size) : Shader(c1) {
  color2 = c2;
  this->size = size;
}

__host__ __device__
Color CheckerShader::shade(Ray& ray, const Light& light,
                           const IntersectionData& data) {
  // example - u = 150, -230
  // -> 1, -3
  int x = floor(data.u / size);
  int y = floor(data.v / size);
  int white = (x + y) % 2;
  Color result = white ? color2 : color;
  result = result * light.color * light.power / 
           (data.p - light.pos).lengthSqr();
  Vector lightDir = light.pos - data.p;
  lightDir.normalize();

  double cosTheta = dot(lightDir, data.normal);
  result = result * cosTheta;
  return result;
}
